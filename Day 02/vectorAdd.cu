#include <iostream>
#include <hip/hip_runtime.h>



__global__ void addVectors(float* dA, float* dB, float* dC, int N){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Even after allocating block size and grid size to match value of N, there are cases where there can be few additional
    // threads allocated (Since block size can only be a multiple of 32), so we need this check
    if(tid < N){
        dC[tid] = dA[tid] + dB[tid];
    }
}


int main(){
    int N = 1 << 20;
    size_t bytes = N * sizeof(float);
    float *hA, *hB, *hC, *dA, *dB, *dC;

    // This allocates pinned memory on the Host side, which is non-pageable and faster
    // For allocating pageable memory, use malloc
    hipHostMalloc(&hA, bytes, hipHostMallocDefault);
    hipHostMalloc(&hB, bytes, hipHostMallocDefault);
    hipHostMalloc(&hC, bytes, hipHostMallocDefault);

    for(int i = 0; i < N; ++i){
        hA[i] = i;
        hB[i] = i + 7;
    }

    // This allocates memory on the devide/CUDA 
    hipMalloc(&dA, bytes);
    hipMalloc(&dB, bytes);
    hipMalloc(&dC, bytes);

    hipMemcpy(dA, hA, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, bytes, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (blockSize + N - 1) / blockSize;

    addVectors<<<gridSize, blockSize>>>(dA, dB, dC, N);

    hipDeviceSynchronize();

    hipMemcpy(hC, dC, bytes, hipMemcpyDeviceToHost);

    for(int i = 0; i < 10; ++i){
        std::cout << hC[i] << std::endl;
    }

    hipFree(dA);
    hipFree(dB); 
    hipFree(dC);

    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);

}